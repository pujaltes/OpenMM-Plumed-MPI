#include "hip/hip_runtime.h"
extern "C" __global__
void addForces(const real* __restrict__ forces, long long* __restrict__ forceBuffers, int* __restrict__ atomIndex) {
    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < NUM_ATOMS; atom += blockDim.x*gridDim.x) {
        int index = atomIndex[atom];
        forceBuffers[atom] += (long long) (forces[3*index]*0x100000000);
        forceBuffers[atom+PADDED_NUM_ATOMS] += (long long) (forces[3*index+1]*0x100000000);
        forceBuffers[atom+2*PADDED_NUM_ATOMS] += (long long) (forces[3*index+2]*0x100000000);
    }
}

